
#include <hip/hip_runtime.h>
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#ifdef WITH_NVSHMEM_SUPPORT
#include <cstddef>
#include <cstdint>
#include <cuda_runtime_api.h>
#include <wholememory/wholememory.h>

#include <wholememory/env_func_ptrs.h>

#include "logger.hpp"
#include "wholememory/communicator.hpp"
#include "wholememory/integer_utils.hpp"
#include "wholememory/memory_handle.hpp"
#include "wholememory_ops/functions/bucket_ids_func.h"
#include "wholememory_ops/functions/exchange_embeddings_nccl_func.h"
#include "wholememory_ops/functions/exchange_ids_nccl_func.h"
#include "wholememory_ops/functions/gather_scatter_func.cuh"
#include "wholememory_ops/functions/gather_scatter_func.h"

#include "wholememory/device_reference.cuh"
#include "wholememory/global_reference.h"
#include "wholememory_ops/register.hpp"
#include "wholememory_ops/temp_memory_handle.hpp"
#include "wholememory_ops/thrust_allocator.hpp"
#include <cub/cub.cuh>
#include <nvshmem.h>
#include <nvshmemx.h>
#include <thrust/sequence.h>
#include <thrust/unique.h>
#include <wholememory/tensor_description.h>
namespace wholememory_ops {

wholememory_error_code_t nvshmem_gather_floating_int32_func(
  wholememory_comm_t wm_comm,
  wholememory_nvshmem_ref_t embeding_nvshmem_ptr,
  wholememory_matrix_description_t embedding_desc,
  const void* indices,
  wholememory_array_description_t indice_desc,
  void* output,
  void* temp_output,
  wholememory_matrix_description_t output_desc,
  size_t embedding_entry_count_per_rank,
  wholememory_env_func_t* p_env_fns,
  cudaStream_t stream);
wholememory_error_code_t nvshmem_gather_floating_int64_func(
  wholememory_comm_t wm_comm,
  wholememory_nvshmem_ref_t embeding_nvshmem_ptr,
  wholememory_matrix_description_t embedding_desc,
  const void* indices,
  wholememory_array_description_t indice_desc,
  void* output,
  void* temp_output,
  wholememory_matrix_description_t output_desc,
  size_t embedding_entry_count_per_rank,
  wholememory_env_func_t* p_env_fns,
  cudaStream_t stream);

wholememory_error_code_t nvshmem_gather_integer_int64_func(
  wholememory_comm_t wm_comm,
  wholememory_nvshmem_ref_t embeding_nvshmem_ptr,
  wholememory_matrix_description_t embedding_desc,
  const void* indices,
  wholememory_array_description_t indice_desc,
  void* output,
  void* temp_output,
  wholememory_matrix_description_t output_desc,
  size_t embedding_entry_count_per_rank,
  wholememory_env_func_t* p_env_fns,
  cudaStream_t stream);
wholememory_error_code_t nvshmem_gather_integer_int32_func(
  wholememory_comm_t wm_comm,
  wholememory_nvshmem_ref_t embeding_nvshmem_ptr,
  wholememory_matrix_description_t embedding_desc,
  const void* indices,
  wholememory_array_description_t indice_desc,
  void* output,
  void* temp_output,
  wholememory_matrix_description_t output_desc,
  size_t embedding_entry_count_per_rank,
  wholememory_env_func_t* p_env_fns,
  cudaStream_t stream);

wholememory_error_code_t wholememory_gather_nvshmem(
  wholememory_handle_t wholememory_handle,
  wholememory_matrix_description_t wholememory_desc,
  void* indices,
  wholememory_array_description_t indice_desc,
  void* output,
  wholememory_matrix_description_t output_desc,
  wholememory_env_func_t* p_env_fns,
  cudaStream_t stream)
{
  try {
    bool embedding_is_float = wholememory_dtype_is_floating_number(wholememory_desc.dtype);
    WHOLEMEMORY_CHECK(embedding_is_float ||
                      wholememory_dtype_is_integer_number(wholememory_desc.dtype));
    bool output_is_float = wholememory_dtype_is_floating_number(output_desc.dtype);
    WHOLEMEMORY_CHECK(output_is_float || wholememory_dtype_is_integer_number(output_desc.dtype));
    WHOLEMEMORY_EXPECTS(
      embedding_is_float == output_is_float,
      "embedding and output should be same number type, e.g. floating number or integer number.");
    if (indice_desc.size == 0) { return WHOLEMEMORY_SUCCESS; }
    size_t embedding_size_per_rank;
    WHOLEMEMORY_RETURN_ON_FAIL(
      wholememory_get_partition_plan(&embedding_size_per_rank, wholememory_handle));

    size_t element_size         = wholememory_dtype_get_element_size(wholememory_desc.dtype);
    size_t embedding_entry_size = element_size * wholememory_desc.stride;

    WHOLEMEMORY_EXPECTS_NOTHROW(
      embedding_size_per_rank % embedding_entry_size == 0,
      "embedding_size_per_rank=%ld is not multiple of embedding_entry_size=%ldx%ld",
      embedding_size_per_rank,
      element_size,
      wholememory_desc.stride);

    size_t embedding_entry_count_per_rank = embedding_size_per_rank / embedding_entry_size;

    wholememory_comm_t wm_comm;
    WHOLEMEMORY_RETURN_ON_FAIL(wholememory_get_communicator(&wm_comm, wholememory_handle));

    wholememory_nvshmem_ref_t embedding_nvshmem_ref;
    WHOLEMEMORY_RETURN_ON_FAIL(
      wholememory_get_nvshmem_reference(&embedding_nvshmem_ref, wholememory_handle));

    temp_memory_handle device_temp_output_handle(p_env_fns);
    size_t temp_output_ele_size = wholememory_get_memory_element_count_from_matrix(&output_desc);
    void* temp_output_ptr =
      device_temp_output_handle.device_malloc(temp_output_ele_size, wholememory_desc.dtype);
    size_t temp_output_byte_size =
      temp_output_ele_size * wholememory_dtype_get_element_size(wholememory_desc.dtype);
    // register
    if (nvshmemx_buffer_register(temp_output_ptr, temp_output_byte_size) != 0) {
      WHOLEMEMORY_ERROR("nvshmemx_buffer_register error in wholememory_gather_nvshmem");
    }

    wholememory_error_code_t (*p_nvshmem_gather_func)(wholememory_comm_t,
                                                      wholememory_nvshmem_ref_t,
                                                      wholememory_matrix_description_t,
                                                      const void*,
                                                      wholememory_array_description_t,
                                                      void*,
                                                      void*,
                                                      wholememory_matrix_description_t,
                                                      size_t,
                                                      wholememory_env_func_t*,
                                                      cudaStream_t) = nullptr;

    if (embedding_is_float) {
      if (indice_desc.dtype == WHOLEMEMORY_DT_INT) {
        p_nvshmem_gather_func = nvshmem_gather_floating_int32_func;
      } else {
        p_nvshmem_gather_func = nvshmem_gather_floating_int64_func;
      }
    } else {
      if (indice_desc.dtype == WHOLEMEMORY_DT_INT) {
        p_nvshmem_gather_func = nvshmem_gather_integer_int32_func;
      } else {
        p_nvshmem_gather_func = nvshmem_gather_integer_int64_func;
      }
    }
    auto ret = p_nvshmem_gather_func(wm_comm,
                                     embedding_nvshmem_ref,
                                     wholememory_desc,
                                     indices,
                                     indice_desc,
                                     output,
                                     temp_output_ptr,
                                     output_desc,
                                     embedding_entry_count_per_rank,
                                     p_env_fns,
                                     stream);
    // ungistre
    WM_CUDA_CHECK(cudaStreamSynchronize(stream));
    if (nvshmemx_buffer_unregister(temp_output_ptr) != 0) {
      WHOLEMEMORY_ERROR("nvshmemx_buffer_unregister error in wholememory_gather_nvshmem");
    }

    WM_CUDA_CHECK(cudaGetLastError());
    return ret;
  } catch (wholememory::cuda_error& wce) {
    WHOLEMEMORY_ERROR("CUDA logic Error %s\n", wce.what());
    return WHOLEMEMORY_CUDA_ERROR;
  } catch (wholememory::logic_error& wle) {
    WHOLEMEMORY_ERROR("LOGIC Error %s\n", wle.what());
    return WHOLEMEMORY_LOGIC_ERROR;
  } catch (...) {
    return WHOLEMEMORY_UNKNOW_ERROR;
  }

  return WHOLEMEMORY_SUCCESS;
}

}  // namespace wholememory_ops

#endif
