/*
 * Copyright (c) 2019-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <hip/hip_runtime_api.h>

#include <wholememory/env_func_ptrs.h>
#include <wholememory/wholememory.h>

#include "logger.hpp"
#include "wholememory/communicator.hpp"
#include "wholememory/memory_handle.hpp"
#include "wholememory_ops/functions/bucket_ids_func.h"
#include "wholememory_ops/functions/exchange_embeddings_nccl_func.h"
#include "wholememory_ops/functions/exchange_ids_nccl_func.h"
#include "wholememory_ops/functions/gather_scatter_func.h"
#include "wholememory_ops/scatter_op_impl.h"
#include "wholememory_ops/temp_memory_handle.hpp"
#include "wholememory_ops/thrust_allocator.hpp"

namespace wholememory_ops {

wholememory_error_code_t wholememory_scatter_nccl(void* input,
                                                  wholememory_matrix_description_t input_desc,
                                                  void* indices,
                                                  wholememory_array_description_t indices_desc,
                                                  wholememory_handle_t wholememory_handle,
                                                  wholememory_matrix_description_t wholememory_desc,
                                                  wholememory_env_func_t* p_env_fns,
                                                  hipStream_t stream,
                                                  int scatter_sms)
{
  try {
    if (wholememory_desc.storage_offset < 0 ||
        wholememory_desc.storage_offset + wholememory_desc.sizes[1] > wholememory_desc.stride) {
      WHOLEMEMORY_ERROR("invalid input offset=%ld, size[1]=%ld, stride=%ld\n",
                        wholememory_desc.storage_offset,
                        wholememory_desc.sizes[1],
                        wholememory_desc.stride);
      return WHOLEMEMORY_INVALID_INPUT;
    }

    wm_thrust_allocator thrust_allocator(p_env_fns);

    size_t element_size         = wholememory_dtype_get_element_size(wholememory_desc.dtype);
    size_t embedding_entry_size = element_size * wholememory_desc.stride;

    wholememory_comm_t wm_comm;
    WHOLEMEMORY_RETURN_ON_FAIL(wholememory_get_communicator(&wm_comm, wholememory_handle));

    int world_size;
    WHOLEMEMORY_RETURN_ON_FAIL(wholememory_communicator_get_size(&world_size, wm_comm));

    temp_memory_handle host_rank_id_count(p_env_fns), host_recv_rank_id_count(p_env_fns);
    int64_t* host_rank_id_count_ptr =
      static_cast<int64_t*>(host_rank_id_count.host_malloc(world_size, WHOLEMEMORY_DT_INT64));
    int64_t* host_recv_rank_id_count_ptr =
      static_cast<int64_t*>(host_recv_rank_id_count.host_malloc(world_size, WHOLEMEMORY_DT_INT64));

    temp_memory_handle dev_recv_indice_buffer(p_env_fns);
    temp_memory_handle dev_raw_indice(p_env_fns);
    int64_t* dev_raw_indice_ptr =
      static_cast<int64_t*>(dev_raw_indice.device_malloc(indices_desc.size, WHOLEMEMORY_DT_INT64));

    int64_t total_recv_count = 0;

    temp_memory_handle dev_embedding_entry_offsets_handle(p_env_fns);
    size_t* dev_embedding_entry_offsets_ptr = static_cast<size_t*>(
      dev_embedding_entry_offsets_handle.device_malloc(world_size + 1, WHOLEMEMORY_DT_INT64));
    temp_memory_handle host_embedding_entry_offsets_handle(p_env_fns);
    size_t* host_embedding_entry_offsets_ptr = static_cast<size_t*>(
      host_embedding_entry_offsets_handle.host_malloc(world_size + 1, WHOLEMEMORY_DT_INT64));

    WHOLEMEMORY_RETURN_ON_FAIL(
      wholememory_get_rank_partition_offsets(host_embedding_entry_offsets_ptr, wholememory_handle));
    for (int i = 0; i < world_size + 1; i++) {
      size_t offset = host_embedding_entry_offsets_ptr[i];
      WHOLEMEMORY_EXPECTS_NOTHROW(
        offset % embedding_entry_size == 0,
        "embedding memory offset of rank%d=%ld is not multiple of embedding_entry_size=%ldx%ld",
        i,
        offset,
        element_size,
        wholememory_desc.stride);
      host_embedding_entry_offsets_ptr[i] /= embedding_entry_size;
    }
    WM_CUDA_CHECK(hipMemcpyAsync(dev_embedding_entry_offsets_ptr,
                                  host_embedding_entry_offsets_ptr,
                                  (world_size + 1) * sizeof(size_t),
                                  hipMemcpyHostToDevice,
                                  stream));
    WHOLEMEMORY_RETURN_ON_FAIL(bucket_and_exchange_ids_func(indices,
                                                            indices_desc,
                                                            host_recv_rank_id_count_ptr,
                                                            host_rank_id_count_ptr,
                                                            &dev_recv_indice_buffer,
                                                            dev_raw_indice_ptr,
                                                            dev_embedding_entry_offsets_ptr,
                                                            wm_comm,
                                                            &thrust_allocator,
                                                            p_env_fns,
                                                            stream));

    // Local Reorder
    for (int i = 0; i < world_size; i++) {
      total_recv_count += host_recv_rank_id_count_ptr[i];
    }
    temp_memory_handle dev_local_reorder_buffer(p_env_fns), dev_embedding_recv_buffer(p_env_fns);
    auto local_reorder_desc =
      wholememory_create_matrix_desc(input_desc.sizes, input_desc.sizes[1], 0, input_desc.dtype);
    void* dev_local_reorder_buffer_ptr = dev_local_reorder_buffer.device_malloc(
      wholememory_get_memory_element_count_from_matrix(&local_reorder_desc), input_desc.dtype);
    wholememory_gref_t input_gref = wholememory_create_continuous_global_reference(input);
    auto dev_raw_indice_desc =
      wholememory_create_array_desc(indices_desc.size, 0, WHOLEMEMORY_DT_INT64);
    WHOLEMEMORY_RETURN_ON_FAIL(gather_func(input_gref,
                                           input_desc,
                                           dev_raw_indice_ptr,
                                           dev_raw_indice_desc,
                                           dev_local_reorder_buffer_ptr,
                                           local_reorder_desc,
                                           stream));
    // AllToAllV for embeddings
    void* dev_embedding_recv_buffer_ptr = dev_embedding_recv_buffer.device_malloc(
      total_recv_count * input_desc.sizes[1], input_desc.dtype);
    size_t embedding_size =
      wholememory_desc.sizes[1] * wholememory_dtype_get_element_size(input_desc.dtype);
    WHOLEMEMORY_RETURN_ON_FAIL(exchange_embeddings_nccl_func(dev_local_reorder_buffer_ptr,
                                                             host_rank_id_count_ptr,
                                                             host_recv_rank_id_count_ptr,
                                                             dev_embedding_recv_buffer_ptr,
                                                             embedding_size,
                                                             wm_comm,
                                                             stream));
    // Local scatter
    size_t local_mem_offset, local_mem_size;
    void* local_fake_ptr = nullptr;
    WHOLEMEMORY_RETURN_ON_FAIL(wholememory_get_local_memory(
      &local_fake_ptr, &local_mem_size, &local_mem_offset, wholememory_handle));
    local_fake_ptr = static_cast<char*>(local_fake_ptr) - local_mem_offset;
    wholememory_gref_t local_fake_embedding_gref =
      wholememory_create_continuous_global_reference(local_fake_ptr);

    std::vector<int64_t> recv_embedding_sizes            = {total_recv_count, input_desc.sizes[1]};
    wholememory_matrix_description_t recv_embedding_desc = wholememory_create_matrix_desc(
      recv_embedding_sizes.data(), input_desc.sizes[1], 0, input_desc.dtype);
    auto recv_indices_desc = wholememory_create_array_desc(total_recv_count, 0, indices_desc.dtype);
    WHOLEMEMORY_RETURN_ON_FAIL(scatter_func(dev_embedding_recv_buffer_ptr,
                                            recv_embedding_desc,
                                            dev_recv_indice_buffer.pointer(),
                                            recv_indices_desc,
                                            local_fake_embedding_gref,
                                            wholememory_desc,
                                            stream,
                                            scatter_sms));
    WM_CUDA_CHECK(hipGetLastError());
    WM_CUDA_CHECK(hipStreamSynchronize(stream));
  } catch (wholememory::cuda_error& wce) {
    WHOLEMEMORY_ERROR("CUDA logic Error %s\n", wce.what());
    return WHOLEMEMORY_CUDA_ERROR;
  } catch (wholememory::logic_error& wle) {
    WHOLEMEMORY_ERROR("LOGIC Error %s\n", wle.what());
    return WHOLEMEMORY_LOGIC_ERROR;
  } catch (...) {
    WHOLEMEMORY_ERROR("Unknown Error\n");
    return WHOLEMEMORY_UNKNOW_ERROR;
  }

  return WHOLEMEMORY_SUCCESS;
}

wholememory_error_code_t wholememory_scatter_distributed(
  void* input,
  wholememory_matrix_description_t input_desc,
  void* indices,
  wholememory_array_description_t indices_desc,
  wholememory_handle_t wholememory_handle,
  wholememory_matrix_description_t wholememory_desc,
  wholememory_env_func_t* p_env_fns,
  hipStream_t stream,
  int scatter_sms)
{
#ifdef WITH_NVSHMEM_SUPPORT
  if (wholememory_get_distributed_backend(wholememory_handle) == WHOLEMEMORY_DB_NVSHMEM) {
    return wholememory_scatter_nvshmem(input,
                                       input_desc,
                                       indices,
                                       indices_desc,
                                       wholememory_handle,
                                       wholememory_desc,
                                       p_env_fns,
                                       stream,
                                       scatter_sms);
  }
#endif

  return wholememory_scatter_nccl(input,
                                  input_desc,
                                  indices,
                                  indices_desc,
                                  wholememory_handle,
                                  wholememory_desc,
                                  p_env_fns,
                                  stream,
                                  scatter_sms);
}
}  // namespace wholememory_ops
