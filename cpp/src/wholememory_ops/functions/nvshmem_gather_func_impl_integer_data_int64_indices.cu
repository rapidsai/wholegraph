
#include <hip/hip_runtime.h>
/*
 * Copyright (c) 2023-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#ifdef WITH_NVSHMEM_SUPPORT

#include "nvshmem_gather_scatter_func.cuh"
#include <wholememory/wholememory.h>

#include "logger.hpp"
#include "wholememory_ops/register.hpp"
namespace wholememory_ops {

template <typename EmbeddingT, typename OutputT>
void nvshmem_gather_integer_int64_temp_func(wholememory_comm_t wm_comm,
                                            wholememory_nvshmem_ref_t embeding_nvshmem_ptr,
                                            wholememory_matrix_description_t embedding_desc,
                                            const void* indices,
                                            int64_t indice_count,
                                            void* output,
                                            void* temp_output,
                                            wholememory_matrix_description_t output_desc,
                                            size_t* embedding_entry_offsets,
                                            wholememory_env_func_t* p_env_fns,
                                            cudaStream_t stream,
                                            int gather_sms)
{
  nvshmem_gather_temp_get_mem_sort_idx_func<EmbeddingT, int64_t, OutputT>(wm_comm,
                                                                          embeding_nvshmem_ptr,
                                                                          embedding_desc,
                                                                          indices,
                                                                          indice_count,
                                                                          output,
                                                                          temp_output,
                                                                          output_desc,
                                                                          embedding_entry_offsets,
                                                                          p_env_fns,
                                                                          stream,
                                                                          gather_sms);
}

REGISTER_DISPATCH_TWO_TYPES(NvshmemGatherFuncIntegerInt64,
                            nvshmem_gather_integer_int64_temp_func,
                            ALLSINT,
                            ALLSINT)

wholememory_error_code_t nvshmem_gather_integer_int64_func(
  wholememory_comm_t wm_comm,
  wholememory_nvshmem_ref_t embeding_nvshmem_ptr,
  wholememory_matrix_description_t embedding_desc,
  const void* indices,
  wholememory_array_description_t indices_desc,
  void* output,
  void* temp_output,
  wholememory_matrix_description_t output_desc,
  size_t* embedding_entry_offsets,
  wholememory_env_func_t* p_env_fns,
  cudaStream_t stream,
  int gather_sms)
{
  try {
    WHOLEMEMORY_CHECK(wholememory_dtype_is_integer_number(embedding_desc.dtype));
    WHOLEMEMORY_CHECK(wholememory_dtype_is_integer_number(output_desc.dtype));
    WHOLEMEMORY_CHECK(indices_desc.dtype == WHOLEMEMORY_DT_INT64);

    DISPATCH_TWO_TYPES(embedding_desc.dtype,
                       output_desc.dtype,
                       NvshmemGatherFuncIntegerInt64,
                       wm_comm,
                       embeding_nvshmem_ptr,
                       embedding_desc,
                       indices,
                       indices_desc.size,
                       output,
                       temp_output,
                       output_desc,
                       embedding_entry_offsets,
                       p_env_fns,
                       stream,
                       gather_sms);
  } catch (const wholememory::cuda_error& wle) {
    WHOLEMEMORY_ERROR("gather CUDA LOGIC Error %s\n", wle.what());
    return WHOLEMEMORY_LOGIC_ERROR;
  } catch (const wholememory::logic_error& le) {
    WHOLEMEMORY_ERROR("gather CUDA LOGIC Error %s\n", le.what());
    return WHOLEMEMORY_LOGIC_ERROR;
  } catch (...) {
    return WHOLEMEMORY_LOGIC_ERROR;
  }
  return WHOLEMEMORY_SUCCESS;
}
template <>
__global__ void scatter_func_with_nvshmem_sort_idxs_kernel<float, int>(
  float* __restrict__ temp_input,
  wholememory_matrix_description_t temp_input_desc,
  wholememory_nvshmem_ref_t embeding_nvshmem_ref,
  wholememory_matrix_description_t embedding_desc,
  const int* __restrict__ sorted_index,  //
  const int* __restrict__ input_index,   //
  int64_t indice_count,
  const int max_blocks_for_local,
  const int intra_node_ranks,
  const int node_rank,
  size_t* embedding_entry_offsets,
  const int threads_per_group);
};  // namespace wholememory_ops

#endif
