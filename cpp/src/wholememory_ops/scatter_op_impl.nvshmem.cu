
#include <hip/hip_runtime.h>
/*
 * Copyright (c) 2023-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#ifdef WITH_NVSHMEM_SUPPORT

#include <cuda_runtime_api.h>
#include <wholememory/wholememory.h>

#include <wholememory/env_func_ptrs.h>

#include "logger.hpp"
#include "wholememory/communicator.hpp"
#include "wholememory/integer_utils.hpp"
#include "wholememory/memory_handle.hpp"
#include "wholememory_ops/functions/bucket_ids_func.h"
#include "wholememory_ops/functions/exchange_embeddings_nccl_func.h"
#include "wholememory_ops/functions/exchange_ids_nccl_func.h"
#include "wholememory_ops/functions/gather_scatter_func.cuh"
#include "wholememory_ops/functions/gather_scatter_func.h"

#include "wholememory_ops/register.hpp"
#include "wholememory_ops/temp_memory_handle.hpp"
#include "wholememory_ops/thrust_allocator.hpp"
#include <wholememory/tensor_description.h>

#include <nvshmem.h>
#include <nvshmemx.h>

namespace wholememory_ops {

wholememory_error_code_t nvshmem_scatter_floating_int32_func(
  wholememory_comm_t wm_comm,
  void* input,
  void* temp_input,
  wholememory_matrix_description_t input_desc,
  const void* indices,
  wholememory_array_description_t indices_desc,
  wholememory_nvshmem_ref_t embeding_nvshmem_ptr,
  wholememory_matrix_description_t embedding_desc,
  size_t* embedding_entry_offsets,
  wholememory_env_func_t* p_env_fns,
  cudaStream_t stream,
  int scatter_sms);

wholememory_error_code_t nvshmem_scatter_floating_int64_func(
  wholememory_comm_t wm_comm,
  void* input,
  void* temp_input,
  wholememory_matrix_description_t input_desc,
  const void* indices,
  wholememory_array_description_t indices_desc,
  wholememory_nvshmem_ref_t embeding_nvshmem_ptr,
  wholememory_matrix_description_t embedding_desc,
  size_t* embedding_entry_offsets,
  wholememory_env_func_t* p_env_fns,
  cudaStream_t stream,
  int scatter_sms);

wholememory_error_code_t nvshmem_scatter_integer_int32_func(
  wholememory_comm_t wm_comm,
  void* input,
  void* temp_input,
  wholememory_matrix_description_t input_desc,
  const void* indices,
  wholememory_array_description_t indices_desc,
  wholememory_nvshmem_ref_t embeding_nvshmem_ptr,
  wholememory_matrix_description_t embedding_desc,
  size_t* embedding_entry_offsets,
  wholememory_env_func_t* p_env_fns,
  cudaStream_t stream,
  int scatter_sms);

wholememory_error_code_t nvshmem_scatter_integer_int64_func(
  wholememory_comm_t wm_comm,
  void* input,
  void* temp_input,
  wholememory_matrix_description_t input_desc,
  const void* indices,
  wholememory_array_description_t indices_desc,
  wholememory_nvshmem_ref_t embeding_nvshmem_ptr,
  wholememory_matrix_description_t embedding_desc,
  size_t* embedding_entry_offsets,
  wholememory_env_func_t* p_env_fns,
  cudaStream_t stream,
  int scatter_sms);

wholememory_error_code_t wholememory_scatter_nvshmem(
  void* input,
  wholememory_matrix_description_t input_desc,
  void* indices,
  wholememory_array_description_t indices_desc,
  wholememory_handle_t wholememory_handle,
  wholememory_matrix_description_t wholememory_desc,
  wholememory_env_func_t* p_env_fns,
  cudaStream_t stream,
  int scatter_sms)
{
  try {
    bool embedding_is_float = wholememory_dtype_is_floating_number(wholememory_desc.dtype);
    WHOLEMEMORY_CHECK(embedding_is_float ||
                      wholememory_dtype_is_integer_number(wholememory_desc.dtype));
    bool input_is_float = wholememory_dtype_is_floating_number(input_desc.dtype);
    WHOLEMEMORY_CHECK(input_is_float || wholememory_dtype_is_integer_number(input_desc.dtype));
    WHOLEMEMORY_EXPECTS(
      embedding_is_float == input_is_float,
      "embedding and output should be same number type, e.g. floating number or integer number.");
    if (indices_desc.size == 0) { return WHOLEMEMORY_SUCCESS; }
    if (wholememory_desc.storage_offset < 0 ||
        wholememory_desc.storage_offset + wholememory_desc.sizes[1] > wholememory_desc.stride) {
      return WHOLEMEMORY_INVALID_INPUT;
    }

    wholememory_comm_t wm_comm;
    WHOLEMEMORY_RETURN_ON_FAIL(wholememory_get_communicator(&wm_comm, wholememory_handle));

    int world_size;
    WHOLEMEMORY_RETURN_ON_FAIL(wholememory_communicator_get_size(&world_size, wm_comm));

    temp_memory_handle dev_embedding_entry_offsets_handle(p_env_fns);
    size_t* dev_embedding_entry_offsets_ptr = static_cast<size_t*>(
      dev_embedding_entry_offsets_handle.device_malloc(world_size + 1, WHOLEMEMORY_DT_INT64));
    temp_memory_handle host_embedding_entry_offsets_handle(p_env_fns);
    size_t* host_embedding_entry_offsets_ptr = static_cast<size_t*>(
      host_embedding_entry_offsets_handle.host_malloc(world_size + 1, WHOLEMEMORY_DT_INT64));

    WHOLEMEMORY_RETURN_ON_FAIL(
      wholememory_get_rank_partition_offsets(host_embedding_entry_offsets_ptr, wholememory_handle));

    size_t element_size         = wholememory_dtype_get_element_size(wholememory_desc.dtype);
    size_t embedding_entry_size = element_size * wholememory_desc.stride;
    for (int i = 0; i < world_size + 1; i++) {
      size_t offset = host_embedding_entry_offsets_ptr[i];
      WHOLEMEMORY_EXPECTS_NOTHROW(
        offset % embedding_entry_size == 0,
        "embedding memory offset of rank%d=%ld is not multiple of embedding_entry_size=%ldx%ld",
        i,
        offset,
        element_size,
        wholememory_desc.stride);
      host_embedding_entry_offsets_ptr[i] /= embedding_entry_size;
    }
    WM_CUDA_CHECK(cudaMemcpyAsync(dev_embedding_entry_offsets_ptr,
                                  host_embedding_entry_offsets_ptr,
                                  (world_size + 1) * sizeof(size_t),
                                  cudaMemcpyHostToDevice,
                                  stream));

    wholememory_nvshmem_ref_t embedding_nvshmem_ref;
    WHOLEMEMORY_RETURN_ON_FAIL(
      wholememory_get_nvshmem_reference(&embedding_nvshmem_ref, wholememory_handle));

    temp_memory_handle device_temp_input_handle(p_env_fns);
    size_t temp_input_ele_size = wholememory_get_memory_element_count_from_matrix(&input_desc);
    void* temp_input_ptr =
      device_temp_input_handle.device_malloc(temp_input_ele_size, wholememory_desc.dtype);
    size_t temp_input_byte_size =
      temp_input_ele_size * wholememory_dtype_get_element_size(wholememory_desc.dtype);
    // register
    if (nvshmemx_buffer_register(temp_input_ptr, temp_input_byte_size) != 0) {
      WHOLEMEMORY_ERROR("nvshmemx_buffer_register error in wholememory_gather_nvshmem");
    }

    wholememory_error_code_t (*p_nvshmem_scatter_func)(wholememory_comm_t,
                                                       void*,
                                                       void*,
                                                       wholememory_matrix_description_t,
                                                       const void*,
                                                       wholememory_array_description_t,
                                                       wholememory_nvshmem_ref_t,
                                                       wholememory_matrix_description_t,
                                                       size_t*,
                                                       wholememory_env_func_t*,
                                                       cudaStream_t,
                                                       int);

    if (embedding_is_float) {
      if (indices_desc.dtype == WHOLEMEMORY_DT_INT) {
        p_nvshmem_scatter_func = nvshmem_scatter_floating_int32_func;
      } else {
        p_nvshmem_scatter_func = nvshmem_scatter_floating_int64_func;
      }
    } else {
      if (indices_desc.dtype == WHOLEMEMORY_DT_INT) {
        p_nvshmem_scatter_func = nvshmem_scatter_integer_int32_func;
      } else {
        p_nvshmem_scatter_func = nvshmem_scatter_integer_int64_func;
      }
    }

    auto ret = p_nvshmem_scatter_func(wm_comm,
                                      input,
                                      temp_input_ptr,
                                      input_desc,
                                      indices,
                                      indices_desc,
                                      embedding_nvshmem_ref,
                                      wholememory_desc,
                                      dev_embedding_entry_offsets_ptr,
                                      p_env_fns,
                                      stream,
                                      scatter_sms);
    if (nvshmemx_buffer_unregister(temp_input_ptr) != 0) {
      WHOLEMEMORY_ERROR("nvshmemx_buffer_unregister error in wholememory_gather_nvshmem");
    }

    WM_CUDA_CHECK(cudaGetLastError());
    return ret;
  } catch (const wholememory::cuda_error& wle) {
    WHOLEMEMORY_ERROR("scatter CUDA LOGIC Error %s\n", wle.what());
    return WHOLEMEMORY_LOGIC_ERROR;
  } catch (const wholememory::logic_error& le) {
    WHOLEMEMORY_ERROR("scatter LOGIC Error %s\n", le.what());
    return WHOLEMEMORY_LOGIC_ERROR;
  } catch (...) {
    return WHOLEMEMORY_UNKNOW_ERROR;
  }
  return WHOLEMEMORY_SUCCESS;
}

}  // namespace wholememory_ops

#endif
