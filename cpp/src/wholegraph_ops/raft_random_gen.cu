#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cmath>
#include <wholememory/wholegraph_op.h>

#include <raft/random/rng_device.cuh>
#include <raft/random/rng_state.hpp>

#include "error.hpp"
#include "logger.hpp"

wholememory_error_code_t generate_random_positive_int_cpu(int64_t random_seed,
                                                          int64_t subsequence,
                                                          wholememory_tensor_t output)
{
  auto output_tensor_desc = *wholememory_tensor_get_tensor_description(output);
  if (output_tensor_desc.dim != 1) {
    WHOLEMEMORY_ERROR("output should be 1D tensor.");
    return WHOLEMEMORY_INVALID_INPUT;
  }
  if (output_tensor_desc.dtype != WHOLEMEMORY_DT_INT64 &&
      output_tensor_desc.dtype != WHOLEMEMORY_DT_INT) {
    WHOLEMEMORY_ERROR("output should be int64 or int32 tensor.");
    return WHOLEMEMORY_INVALID_INPUT;
  }

  auto* output_ptr = wholememory_tensor_get_data_pointer(output);

  raft::random::RngState _rngstate(random_seed, 0, raft::random::GeneratorType::GenPC);
  raft::random::detail::DeviceState<raft::random::detail::PCGenerator> rngstate(_rngstate);
  raft::random::detail::PCGenerator rng(rngstate, (uint64_t)subsequence);

  for (int64_t i = 0; i < output_tensor_desc.sizes[0]; i++) {
    if (output_tensor_desc.dtype == WHOLEMEMORY_DT_INT) {
      raft::random::detail::UniformDistParams<int32_t> params;
      params.start = 0;
      params.end   = 1;
      int32_t random_num;
      raft::random::detail::custom_next(rng, &random_num, params, 0, 0);
      static_cast<int*>(output_ptr)[i] = random_num;
    } else {
      raft::random::detail::UniformDistParams<int64_t> params;
      params.start = 0;
      params.end   = 1;
      int64_t random_num;
      raft::random::detail::custom_next(rng, &random_num, params, 0, 0);
      static_cast<int64_t*>(output_ptr)[i] = random_num;
    }
  }
  return WHOLEMEMORY_SUCCESS;
}

wholememory_error_code_t generate_exponential_distribution_negative_float_cpu(
  int64_t random_seed, int64_t subsequence, wholememory_tensor_t output)
{
  auto output_tensor_desc = *wholememory_tensor_get_tensor_description(output);
  if (output_tensor_desc.dim != 1) {
    WHOLEMEMORY_ERROR("output should be 1D tensor.");
    return WHOLEMEMORY_INVALID_INPUT;
  }
  if (output_tensor_desc.dtype != WHOLEMEMORY_DT_FLOAT) {
    WHOLEMEMORY_ERROR("output should be float.");
    return WHOLEMEMORY_INVALID_INPUT;
  }
  auto* output_ptr = wholememory_tensor_get_data_pointer(output);
  raft::random::RngState _rngstate(random_seed, 0, raft::random::GeneratorType::GenPC);
  raft::random::detail::DeviceState<raft::random::detail::PCGenerator> rngstate(_rngstate);
  raft::random::detail::PCGenerator rng(rngstate, (uint64_t)subsequence);
  for (int64_t i = 0; i < output_tensor_desc.sizes[0]; i++) {
    float u = 0.0;
    rng.next(u);
    u                    = -(0.5 + 0.5 * u);
    uint64_t random_num2 = 0;
    int seed_count       = -1;
    do {
      rng.next(random_num2);
      seed_count++;
    } while (!random_num2);
    auto count_one = [](unsigned long long num) {
      int32_t c = 0;
      while (num) {
        num >>= 1;
        c++;
      }
      return 64 - c;
    };
    int32_t one_bit = count_one(random_num2) + seed_count * 64;
    u *= pow(2, -one_bit);
    // float logk = (log1pf(u) / logf(2.0)) * (1.0f / (float)weight);
    float logk                         = (log1p(u) / log(2.0));
    static_cast<float*>(output_ptr)[i] = logk;
  }
  return WHOLEMEMORY_SUCCESS;
}
