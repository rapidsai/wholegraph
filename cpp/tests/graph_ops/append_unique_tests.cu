/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cstdint>
#include <cstdio>
#include <gtest/gtest.h>

#include "../wholegraph_ops/graph_sampling_test_utils.hpp"
#include "../wholememory/wholememory_test_utils.hpp"
#include "append_unique_test_utils.hpp"
#include "parallel_utils.hpp"
#include "wholememory/communicator.hpp"
#include "wholememory/env_func_ptrs.hpp"
#include "wholememory/initialize.hpp"
#include <wholememory/graph_op.h>
#include <wholememory/tensor_description.h>

typedef struct GraphAppendUniqueTestParam {
  GraphAppendUniqueTestParam& set_target_node_count(int new_target_node_count)
  {
    target_node_count = new_target_node_count;
    return *this;
  }
  GraphAppendUniqueTestParam& set_neighbor_node_count(int new_neighbor_node_count)
  {
    neighbor_node_count = new_neighbor_node_count;
    return *this;
  }

  GraphAppendUniqueTestParam& set_target_dtype(wholememory_dtype_t new_target_node_dtype)
  {
    target_node_dtype   = new_target_node_dtype;
    neighbor_node_dtype = new_target_node_dtype;
    return *this;
  }
  wholememory_array_description_t get_target_node_desc() const
  {
    return wholememory_create_array_desc(target_node_count, 0, target_node_dtype);
  }
  wholememory_array_description_t get_neighbor_node_desc() const
  {
    return wholememory_create_array_desc(neighbor_node_count, 0, neighbor_node_dtype);
  }

  int64_t get_target_node_count() const { return target_node_count; }
  int64_t get_neighbor_node_count() const { return neighbor_node_count; }
  wholememory_dtype_t target_node_dtype   = WHOLEMEMORY_DT_INT;
  wholememory_dtype_t neighbor_node_dtype = target_node_dtype;
  int64_t target_node_count               = 10;
  int64_t neighbor_node_count             = 100;
} GraphAppendUniqueTestParam;

class GraphAppendUniqueParameterTests
  : public ::testing::TestWithParam<GraphAppendUniqueTestParam> {};

TEST_P(GraphAppendUniqueParameterTests, AppendUniqueTest)
{
  auto params = GetParam();
  int dev_count;
  EXPECT_EQ(hipGetDeviceCount(&dev_count), hipSuccess);
  EXPECT_GE(dev_count, 1);

  hipStream_t stream;
  EXPECT_EQ(hipStreamCreate(&stream), hipSuccess);
  auto target_node_count   = params.get_target_node_count();
  auto neighbor_node_count = params.get_neighbor_node_count();
  auto target_node_desc    = params.get_target_node_desc();
  auto neighbor_node_desc  = params.get_neighbor_node_desc();

  size_t target_node_size   = wholememory_get_memory_size_from_array(&target_node_desc);
  size_t neighbor_node_size = wholememory_get_memory_size_from_array(&neighbor_node_desc);

  void *host_target_nodes_ptr = nullptr, *host_neighbor_nodes_ptr = nullptr;
  void *dev_target_nodes_ptr = nullptr, *dev_neighbor_nodes_ptr = nullptr;
  void *host_output_unique_nodes_ptr = nullptr, *ref_host_output_unique_nodes_ptr = nullptr;
  int *host_output_neighbor_raw_to_unique_mapping_ptr     = nullptr,
      *ref_host_output_neighbor_raw_to_unique_mapping_ptr = nullptr;
  int* dev_output_neighbor_raw_to_unique_mapping_ptr      = nullptr;
  wholememory_array_description_t neighbor_raw_to_unique_mapping_desc =
    wholememory_create_array_desc(neighbor_node_count, 0, WHOLEMEMORY_DT_INT);

  EXPECT_EQ(hipHostMalloc(&host_target_nodes_ptr, target_node_size), hipSuccess);
  EXPECT_EQ(hipHostMalloc(&host_neighbor_nodes_ptr, neighbor_node_size), hipSuccess);
  EXPECT_EQ(hipMalloc(&dev_target_nodes_ptr, target_node_size), hipSuccess);
  EXPECT_EQ(hipMalloc(&dev_neighbor_nodes_ptr, neighbor_node_size), hipSuccess);
  EXPECT_EQ(
    hipMalloc(&dev_output_neighbor_raw_to_unique_mapping_ptr, neighbor_node_count * sizeof(int)),
    hipSuccess);
  int64_t total_node_count = neighbor_node_count + target_node_count;
  graph_ops::testing::gen_node_ids(host_target_nodes_ptr, target_node_desc, total_node_count, true);
  graph_ops::testing::gen_node_ids(
    host_neighbor_nodes_ptr, neighbor_node_desc, total_node_count, false);

  EXPECT_EQ(hipMemcpyAsync(dev_target_nodes_ptr,
                            host_target_nodes_ptr,
                            target_node_size,
                            hipMemcpyHostToDevice,
                            stream),
            hipSuccess);
  EXPECT_EQ(hipMemcpyAsync(dev_neighbor_nodes_ptr,
                            host_neighbor_nodes_ptr,
                            neighbor_node_size,
                            hipMemcpyHostToDevice,
                            stream),
            hipSuccess);
  wholememory_tensor_t target_node_tensor, neighbor_node_tensor,
    output_neighbor_raw_to_unique_mapping_tensor;
  wholememory_tensor_description_t target_node_tensor_desc, neighbor_node_tensor_desc,
    output_neighbor_raw_to_unique_mapping_tensor_desc;
  wholememory_copy_array_desc_to_tensor(&target_node_tensor_desc, &target_node_desc);
  wholememory_copy_array_desc_to_tensor(&neighbor_node_tensor_desc, &neighbor_node_desc);
  wholememory_copy_array_desc_to_tensor(&output_neighbor_raw_to_unique_mapping_tensor_desc,
                                        &neighbor_raw_to_unique_mapping_desc);
  EXPECT_EQ(wholememory_make_tensor_from_pointer(
              &target_node_tensor, dev_target_nodes_ptr, &target_node_tensor_desc),
            WHOLEMEMORY_SUCCESS);
  EXPECT_EQ(wholememory_make_tensor_from_pointer(
              &neighbor_node_tensor, dev_neighbor_nodes_ptr, &neighbor_node_tensor_desc),
            WHOLEMEMORY_SUCCESS);
  EXPECT_EQ(
    wholememory_make_tensor_from_pointer(&output_neighbor_raw_to_unique_mapping_tensor,
                                         dev_output_neighbor_raw_to_unique_mapping_ptr,
                                         &output_neighbor_raw_to_unique_mapping_tensor_desc),
    WHOLEMEMORY_SUCCESS);
  wholememory_env_func_t* default_env_func = wholememory::get_default_env_func();
  wholememory::default_memory_context_t output_unique_node_memory_ctx;
  EXPECT_EQ(graph_append_unique(target_node_tensor,
                                neighbor_node_tensor,
                                &output_unique_node_memory_ctx,
                                output_neighbor_raw_to_unique_mapping_tensor,
                                default_env_func,
                                stream),
            WHOLEMEMORY_SUCCESS);
  EXPECT_EQ(hipGetLastError(), hipSuccess);
  EXPECT_EQ(hipStreamSynchronize(stream), hipSuccess);

  int total_unique_count = output_unique_node_memory_ctx.desc.sizes[0];

  host_output_unique_nodes_ptr =
    malloc(total_unique_count * wholememory_dtype_get_element_size(target_node_desc.dtype));
  host_output_neighbor_raw_to_unique_mapping_ptr = (int*)malloc(neighbor_node_count * sizeof(int));
  EXPECT_EQ(
    hipMemcpyAsync(host_output_unique_nodes_ptr,
                    output_unique_node_memory_ctx.ptr,
                    total_unique_count * wholememory_dtype_get_element_size(target_node_desc.dtype),
                    hipMemcpyDeviceToHost,
                    stream),
    hipSuccess);
  EXPECT_EQ(hipMemcpyAsync(host_output_neighbor_raw_to_unique_mapping_ptr,
                            dev_output_neighbor_raw_to_unique_mapping_ptr,
                            neighbor_node_count * sizeof(int),
                            hipMemcpyDeviceToHost,
                            stream),
            hipSuccess);
  EXPECT_EQ(hipStreamSynchronize(stream), hipSuccess);

  int ref_total_unique_node_count;
  graph_ops::testing::host_append_unique(
    host_target_nodes_ptr,
    target_node_desc,
    host_neighbor_nodes_ptr,
    neighbor_node_desc,
    &ref_total_unique_node_count,
    &ref_host_output_unique_nodes_ptr);

  EXPECT_EQ(total_unique_count, ref_total_unique_node_count);
  graph_ops::testing::host_gen_append_unique_neighbor_raw_to_unique(host_output_unique_nodes_ptr, wholememory_create_array_desc(total_unique_count, 0, target_node_desc.dtype),
    host_neighbor_nodes_ptr,
    neighbor_node_desc,
    (void**)&ref_host_output_neighbor_raw_to_unique_mapping_ptr,
    neighbor_raw_to_unique_mapping_desc);
  
  if (target_node_desc.dtype == WHOLEMEMORY_DT_INT) {
    std::sort(static_cast<int*>(host_output_unique_nodes_ptr) + target_node_count, static_cast<int*>(host_output_unique_nodes_ptr) + total_unique_count);
    std::sort(static_cast<int*>(ref_host_output_unique_nodes_ptr) + target_node_count, static_cast<int*>(ref_host_output_unique_nodes_ptr) + total_unique_count);
  }
  else if (target_node_desc.dtype == WHOLEMEMORY_DT_INT64) {
    std::sort(static_cast<int64_t*>(host_output_unique_nodes_ptr) + target_node_count, static_cast<int64_t*>(host_output_unique_nodes_ptr) + total_unique_count);
    std::sort(static_cast<int64_t*>(ref_host_output_unique_nodes_ptr) + target_node_count, static_cast<int64_t*>(ref_host_output_unique_nodes_ptr) + total_unique_count);
  }

  wholegraph_ops::testing::host_check_two_array_same(
    host_output_unique_nodes_ptr,
    wholememory_create_array_desc(total_unique_count, 0, target_node_desc.dtype),
    ref_host_output_unique_nodes_ptr,
    wholememory_create_array_desc(ref_total_unique_node_count, 0, target_node_desc.dtype));

  wholegraph_ops::testing::host_check_two_array_same(
    host_output_neighbor_raw_to_unique_mapping_ptr,
    neighbor_raw_to_unique_mapping_desc,
    ref_host_output_neighbor_raw_to_unique_mapping_ptr,
    neighbor_raw_to_unique_mapping_desc);

  (default_env_func->output_fns).free_fn(&output_unique_node_memory_ctx, nullptr);
  if (host_output_unique_nodes_ptr != nullptr) { free(host_output_unique_nodes_ptr); }
  if (host_output_neighbor_raw_to_unique_mapping_ptr != nullptr) {
    free(host_output_neighbor_raw_to_unique_mapping_ptr);
  }
  if (ref_host_output_unique_nodes_ptr != nullptr) { free(ref_host_output_unique_nodes_ptr); }
  if (ref_host_output_neighbor_raw_to_unique_mapping_ptr != nullptr) {
    free(ref_host_output_neighbor_raw_to_unique_mapping_ptr);
  }

  EXPECT_EQ(hipHostFree(host_target_nodes_ptr), hipSuccess);
  EXPECT_EQ(hipHostFree(host_neighbor_nodes_ptr), hipSuccess);
  EXPECT_EQ(hipFree(dev_target_nodes_ptr), hipSuccess);
  EXPECT_EQ(hipFree(dev_neighbor_nodes_ptr), hipSuccess);
  EXPECT_EQ(hipFree(dev_output_neighbor_raw_to_unique_mapping_ptr), hipSuccess);

  WHOLEMEMORY_CHECK(::testing::Test::HasFailure() == false);
}

INSTANTIATE_TEST_SUITE_P(GraphAppendUniqueOpTests,
                         GraphAppendUniqueParameterTests,
                         ::testing::Values(GraphAppendUniqueTestParam()
                                             .set_target_node_count(3)
                                             .set_neighbor_node_count(10),
                                           GraphAppendUniqueTestParam()
                                             .set_target_node_count(53)
                                             .set_neighbor_node_count(123)
                                             .set_target_dtype(WHOLEMEMORY_DT_INT),
                                           GraphAppendUniqueTestParam()
                                             .set_target_node_count(57)
                                             .set_neighbor_node_count(1235)
                                             .set_target_dtype(WHOLEMEMORY_DT_INT64)));
